#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hip/hip_complex.h>
#include <iostream>
#include <vector>
#include <random>
#include <cmath>
#include <iomanip>

#define CHECK_CUDA(call) do { \
    hipError_t err = call; \
    if (err != hipSuccess) { \
        std::cerr << "CUDA error at " << __FILE__ << ":" << __LINE__ << " - " << hipGetErrorString(err) << std::endl; \
        exit(1); \
    } \
} while(0)

#define CHECK_CUBLAS(call) do { \
    hipblasStatus_t status = call; \
    if (status != HIPBLAS_STATUS_SUCCESS) { \
        std::cerr << "cuBLAS error at " << __FILE__ << ":" << __LINE__ << " - " << status << std::endl; \
        exit(1); \
    } \
} while(0)

template<typename T>
__device__ __forceinline__ T make_zero() {
    return make_hipDoubleComplex(0.0, 0.0);
}

template<>
__device__ __forceinline__ hipDoubleComplex make_zero<hipDoubleComplex>() {
    return make_hipDoubleComplex(0.0, 0.0);
}

template<typename T>
__device__ __forceinline__ T conjugate(T x) {
    return hipConj(x);
}

template<typename T>
__device__ __forceinline__ T make_real(T x) {
    return make_hipDoubleComplex(hipCreal(x), 0.0);
}

// Custom HEMV kernel based on the provided template
template <class T, int syhemv_bs, int thread_x, int thread_y, int elements_per_thread>
__global__ void
syhemvl_generic_d( int n, T alpha,
				    T *A, int lda,
				    T *x, int incx,
				    T  beta,
				    T *y, int incy,
				    int	    n_mod_syhemv_bs)
{
    const int tx   = threadIdx.x ;
    const int ty   = threadIdx.y ;
    const int blkc = blockIdx.x ;
    const int td  = (thread_x * ty ) + tx;

    T res  = make_zero<T>();
    T yold = make_zero<T>();

    __shared__ T la   [syhemv_bs * syhemv_bs];
    __shared__ T buff [syhemv_bs];
    __shared__ T accum[syhemv_bs * (2 * thread_y)];

	// Advance 'A' to start of diagonal blocks first
	A += syhemv_bs * blkc * (lda + 1);

	// Advance 'A' to start row for each thread inside the diagonal block
	A += ty * lda + tx;

	// Advance x
	x += (blkc * syhemv_bs) * incx;

	// Advance y
	y += (blkc * syhemv_bs) * incy;

	// load part of vector x
	if(blkc == gridDim.x-1)
	{
		if(ty == 0)
		{
	  		if(tx < n_mod_syhemv_bs)
	  		{
	    		buff[tx] = x[incx * tx];
	    		yold = hipCmul(beta, y[tx * incy]);
	    	}
	    	else
	    	{
	    		buff[tx] = make_zero<T>();
	    		yold = make_zero<T>();
	    	}
	  	}
	}
	else
	{
	  	if(ty == 0)
	  	{
			buff[tx] = x[incx * tx];
			yold = hipCmul(beta, y[tx * incy]);
		}
	} // end of load part of vector x

	// init shmem (last TB only)
	if(blkc == gridDim.x-1)
	{
		#pragma unroll
		for(int j = 0; j < syhemv_bs; j+= thread_y)
			la[j * syhemv_bs + td ] = make_zero<T>();
		__syncthreads();

		if(tx >= n_mod_syhemv_bs) return; 	// these threads should not read any useful data
	}

	// load a block of data
	if(blkc == gridDim.x-1)
	{
		int j;
		#pragma unroll
		for(j = 0; j < n_mod_syhemv_bs/thread_y; j++)
			la[(j*thread_y) * syhemv_bs + td] = A[(j*thread_y) * lda];

		if(ty < (n_mod_syhemv_bs%thread_y))
			la[(j*thread_y) * syhemv_bs + td] = A[(j*thread_y) * lda];
	}
	else
	{
		#pragma unroll
		for(int j = 0; j < syhemv_bs; j+= thread_y)
			la[j * syhemv_bs + td] = A[j * lda];
	}
	// end of reading a diagonal block of data

	__syncthreads();

	// mirror necessary elements in first chunk
	if(ty > tx)
		la[td] = conjugate( la[tx * syhemv_bs + ty] );
	else
		la[td] = la[td];

	#pragma unroll
	for(int j = thread_y; j < (syhemv_bs/2); j+= thread_y)
		if(abs(tx - ty) < j)
			la[tx + (ty + j) * syhemv_bs] = conjugate( la[ty + j + tx * syhemv_bs] );

	// mirror second chunk
	#pragma unroll
	for(int j = 0; j < (syhemv_bs/2); j+= thread_y)
		if(abs(tx-ty) < (j + (syhemv_bs/2)))
			la[syhemv_bs * ((syhemv_bs/2) + j + ty) + tx] = conjugate( la[syhemv_bs * tx + (syhemv_bs/2) + j + ty] );

	// ignore imaginary part of diagonal elements
	if(ty == 0) la[tx * syhemv_bs + tx] = make_real(la[tx * syhemv_bs + tx]);

	__syncthreads();

	// compute first chunk
	#pragma unroll
	for(int j = 0; j < (syhemv_bs/2); j+= thread_y)
		res = hipCadd(res, hipCmul(la[(ty + j) * syhemv_bs + tx], buff[j + ty]));

	// compute second chunk
	#pragma unroll
	for(int j = (syhemv_bs/2); j < 2 * (syhemv_bs/2); j+= thread_y)
		res = hipCadd(res, hipCmul(la[(ty + j) * syhemv_bs + tx], buff[j + ty]));

	accum[td] = res;
	__syncthreads();
	if(ty == 0)
	{
		res = make_zero<T>();
	  	#pragma unroll
	  	for(int j = 0; j < thread_y; j++)
			res = hipCadd(res, accum[j * syhemv_bs + tx]);
	  	res = hipCmul(alpha, res);
	  	res = hipCadd(res, yold);
	  	if(blkc == gridDim.x-1){if(tx < n_mod_syhemv_bs)y[tx * incy] = res;}
	  	else{y[tx * incy] = res;}
	}
}

template <class T, int syhemv_bs, int thread_x, int thread_y, int elements_per_thread >
__global__ void
syhemvl_generic_nd( int n, T alpha,
                               T *A, int lda,
                               T *x, int incx,
                               T  beta,
                               T *y, int incy,
								int     n_mod_syhemv_bs)
{
    const int tx   = threadIdx.x ;
    const int ty   = threadIdx.y ;
    const int blkc = blockIdx.x ;
    const int by 	= blockIdx.y;
    const int td  = (thread_x * ty ) + tx;
    const int tx_  = td % (syhemv_bs/2);
    const int ty_  = td / (syhemv_bs/2);
    T *xcopy, *ycopy;

    int count = (gridDim.x-blkc-1-1)/gridDim.y;

    T xreg[elements_per_thread];
    T areg[elements_per_thread];
    T treg[elements_per_thread] = {make_zero<T>()};

    T res_1_	= make_zero<T>();
    T res_2_	= make_zero<T>();
    T x1		= make_zero<T>();
    T x2		= make_zero<T>();

    __shared__ T la   [syhemv_bs * (syhemv_bs/2)];
    __shared__ T accum[syhemv_bs * (2 * thread_y)];
    __shared__ T xbuff[syhemv_bs];

    if(blkc == gridDim.x - 1)return;

    // Advance 'A' to start of diagonal blocks first
    A += syhemv_bs * blkc * (lda + 1);
    // divide work among the y-direction of the grid
	A += (by * count) * syhemv_bs;

    // Advance 'x'
    x += (blkc * syhemv_bs) * incx;
    xcopy = x;
    x += (by * count * syhemv_bs) * incx;

    if(ty == 0) xbuff[tx] = xcopy[incx * tx];

    //Advance 'y'
	y += (blkc * syhemv_bs) * incy;
    ycopy = y;
    ycopy += (by * count * syhemv_bs) * incy;

    if(by == gridDim.y-1) count += ((gridDim.x-blkc-1-1)%gridDim.y);
    if(by != gridDim.y-1){if(count == 0) return;}

	int j = ty_ * elements_per_thread * lda + tx_;

	__syncthreads();

    A += syhemv_bs;
    x += syhemv_bs * incx;

    if(blkc < gridDim.x-2)		// to prevent out of bound access
    {
    	#pragma unroll
    	for(int k = 0; k < elements_per_thread; k++)
			xreg[k] = A[j + k * lda];
    	x1 = x[incx * tx_];
    }

    A -= syhemv_bs;
    x -= syhemv_bs * incx;

    #pragma unroll
    for(int Vblocks = 0; Vblocks < count; Vblocks++)
    {
		A += syhemv_bs;
		x += syhemv_bs * incx;

		res_1_ = make_zero<T>();
		res_2_ = make_zero<T>();

		x2 = x[incx * (tx_ + (syhemv_bs/2))];

		#pragma unroll
		for(int k = 0; k < elements_per_thread; k++)
	    	areg[k] = A[(syhemv_bs/2) + j + k * lda];

		#pragma unroll
		for(int k = 0; k < elements_per_thread; k++)
		{
	    	res_1_ = hipCadd(res_1_, hipCmul(xreg[k], xbuff[ty_ * elements_per_thread + k]));
	    	treg[k] = hipCadd(treg[k], hipCmul(conjugate(xreg[k]), x1));
		}

		A += syhemv_bs;
		x += syhemv_bs * incx;

		if(Vblocks != count-1)
		{
			#pragma unroll
			for(int k = 0; k < elements_per_thread; k++)
	  			xreg[k] = A[j + k * lda];
	  		x1 = x[incx * tx_];
	  	}

		A -= syhemv_bs;
		x -= syhemv_bs * incx;

		#pragma unroll
		for(int k = 0; k < elements_per_thread; k++)
		{
	  		res_2_ = hipCadd(res_2_, hipCmul(areg[k], xbuff[ty_ * elements_per_thread + k]));
	  		treg[k] = hipCadd(treg[k], hipCmul(conjugate(areg[k]), x2));
		}

		// Horizontal block should be stored in global memory
		__syncthreads();
		accum[ty_ * syhemv_bs + tx_] = res_1_;
		accum[ty_ * syhemv_bs + tx_ + (syhemv_bs/2)] = res_2_;
		__syncthreads();
		if(ty == 0)
		{
			ycopy += syhemv_bs * incy;
	   		res_1_ = make_zero<T>();
	   		#pragma unroll
	   		for(int k = 0; k < (2 * thread_y); k++)
	    		res_1_ = hipCadd(res_1_, accum[k * syhemv_bs + tx]);

	    	res_1_ = hipCmul(alpha, res_1_);
	    	// use atomics for complex numbers
	    	T old_val, new_val;
	    	do {
	    		old_val = ycopy[incy * tx];
	    		new_val = hipCadd(old_val, res_1_);
	    	} while (atomicCAS((unsigned long long*)&ycopy[incy * tx], 
	    	                   __double_as_longlong(hipCreal(old_val)) | 
	    	                   ((unsigned long long)__double_as_longlong(hipCimag(old_val)) << 32),
	    	                   __double_as_longlong(hipCreal(new_val)) | 
	    	                   ((unsigned long long)__double_as_longlong(hipCimag(new_val)) << 32)) != 
	    	         (__double_as_longlong(hipCreal(old_val)) | 
	    	          ((unsigned long long)__double_as_longlong(hipCimag(old_val)) << 32)));
		}
    }// end of for loop on blocks

    //////////////////////////////////////////////////
    // last irregular tile
    if(by == gridDim.y-1)
    {
    	res_1_ = make_zero<T>();
    	res_2_ = make_zero<T>();

		A += syhemv_bs;
		x += syhemv_bs * incx;

    	#pragma unroll
    	for(int k = 0; k < elements_per_thread; k++)
    	{
    		xreg[k] = make_zero<T>();
    		areg[k] = make_zero<T>();
    	}

    	if(tx_ < n_mod_syhemv_bs)
    	{
			#pragma unroll
			for(int k = 0; k < elements_per_thread; k++)
				xreg[k] = A[j + k * lda];

			x1 = x[incx * tx_];
		}

		if( (tx_+(syhemv_bs/2)) < n_mod_syhemv_bs)
		{
			#pragma unroll
    		for(int k = 0; k < elements_per_thread; k++)
				areg[k] = A[(syhemv_bs/2) + j + k * lda];

			x2 = x[incx * (tx_ + (syhemv_bs/2))];
		}

    	#pragma unroll
    	for(int k = 0; k < elements_per_thread; k++)
    	{
			res_1_ = hipCadd(res_1_, hipCmul(xreg[k], xbuff[ty_ * elements_per_thread + k]));
			treg[k] = hipCadd(treg[k], hipCmul(conjugate(xreg[k]), x1));
		}

		#pragma unroll
		for(int k = 0; k < elements_per_thread; k++)
		{
			res_2_ = hipCadd(res_2_, hipCmul(areg[k], xbuff[ty_ * elements_per_thread + k]));
			treg[k] = hipCadd(treg[k], hipCmul(conjugate(areg[k]), x2));
		}

    	// Horizontal block reduction
		__syncthreads();
		accum[ty_ * syhemv_bs + tx_] = res_1_;
		accum[ty_ * syhemv_bs + tx_ + (syhemv_bs/2)] = res_2_;
		__syncthreads();
		if(ty == 0)
		{
			ycopy += syhemv_bs * incy;
	   		res_1_ = make_zero<T>();
	   		#pragma unroll
	   		for(int k = 0; k < (2 * thread_y); k++)
	    		res_1_ = hipCadd(res_1_, accum[k * syhemv_bs + tx]);

	    	res_1_ = hipCmul(alpha, res_1_);
	    	// use atomics
	    	if(tx < n_mod_syhemv_bs) {
	    		T old_val, new_val;
		    	do {
		    		old_val = ycopy[incy * tx];
		    		new_val = hipCadd(old_val, res_1_);
		    	} while (atomicCAS((unsigned long long*)&ycopy[incy * tx], 
		    	                   __double_as_longlong(hipCreal(old_val)) | 
		    	                   ((unsigned long long)__double_as_longlong(hipCimag(old_val)) << 32),
		    	                   __double_as_longlong(hipCreal(new_val)) | 
		    	                   ((unsigned long long)__double_as_longlong(hipCimag(new_val)) << 32)) != 
		    	         (__double_as_longlong(hipCreal(old_val)) | 
		    	          ((unsigned long long)__double_as_longlong(hipCimag(old_val)) << 32)));
	    	}
		}
	}

	#pragma unroll
    for(int k = 0; k < elements_per_thread; k++)
    	la[(ty_ * elements_per_thread + k) * (syhemv_bs/2) + tx_] = treg[k];

    __syncthreads();		// important

    if(ty == 0)
    {
		treg[0] = make_zero<T>(); // tmp accumulator
		#pragma unroll
		for(int j = tx; j < tx+(syhemv_bs/2); j++)
	  		treg[0] = hipCadd(treg[0], la[tx * (syhemv_bs/2) +  (j % (syhemv_bs/2))]);

	   	treg[0] = hipCmul(alpha, treg[0]);
	   	atomicAddComplex(&y[incy * tx], treg[0]);
	}
}

// Error checking function equivalent to torch.allclose for complex numbers
bool compareResults(const hipDoubleComplex* a, const hipDoubleComplex* b, int size, double rtol = 1e-5, double atol = 1e-8) {
    double max_error = 0.0;
    int max_error_index = -1;
    int first_mismatch_index = -1;
    int mismatch_count = 0;
    
    for (int i = 0; i < size; i++) {
        double diff_real = std::abs(hipCreal(a[i]) - hipCreal(b[i]));
        double diff_imag = std::abs(hipCimag(a[i]) - hipCimag(b[i]));
        double diff = std::sqrt(diff_real * diff_real + diff_imag * diff_imag);
        
        double tolerance_real = atol + rtol * std::max(std::abs(hipCreal(a[i])), std::abs(hipCreal(b[i])));
        double tolerance_imag = atol + rtol * std::max(std::abs(hipCimag(a[i])), std::abs(hipCimag(b[i])));
        double tolerance = std::sqrt(tolerance_real * tolerance_real + tolerance_imag * tolerance_imag);
        
        if (diff > max_error) {
            max_error = diff;
            max_error_index = i;
        }
        
        if (diff > tolerance) {
            if (first_mismatch_index == -1) {
                first_mismatch_index = i;
            }
            mismatch_count++;
            
            // Print first few mismatches for debugging
            if (mismatch_count <= 5) {
                std::cout << "Mismatch at index " << i << ": " 
                         << "a[" << i << "] = " << std::scientific << hipCreal(a[i]) << "+" << hipCimag(a[i]) << "i"
                         << ", b[" << i << "] = " << std::scientific << hipCreal(b[i]) << "+" << hipCimag(b[i]) << "i"
                         << ", diff = " << std::scientific << diff 
                         << ", tolerance = " << std::scientific << tolerance << std::endl;
            }
        }
    }
    
    std::cout << "Maximum error between results: " << std::scientific << max_error;
    if (max_error_index != -1) {
        std::cout << " (at index " << max_error_index << ")";
    }
    std::cout << std::endl;
    
    if (mismatch_count > 0) {
        std::cout << "Total mismatches: " << mismatch_count << " out of " << size << " elements" << std::endl;
        if (mismatch_count > 5) {
            std::cout << "(Only showing first 5 mismatches)" << std::endl;
        }
        return false;
    }
    
    return true;
}

void initializeHermitianMatrix(hipDoubleComplex* matrix, int n) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dist(-1.0, 1.0);
    
    // Initialize matrix with random values and ensure Hermitian property
    for (int i = 0; i < n; i++) {
        for (int j = 0; j < n; j++) {
            if (i <= j) {
                if (i == j) {
                    // Diagonal elements must be real for Hermitian matrix
                    matrix[i * n + j] = make_hipDoubleComplex(dist(gen), 0.0);
                } else {
                    // Off-diagonal elements
                    matrix[i * n + j] = make_hipDoubleComplex(dist(gen), dist(gen));
                    // Ensure Hermitian property: A[j,i] = conj(A[i,j])
                    matrix[j * n + i] = hipConj(matrix[i * n + j]);
                }
            }
        }
    }
}

void initializeComplexVector(hipDoubleComplex* vector, int size) {
    std::random_device rd;
    std::mt19937 gen(rd());
    std::uniform_real_distribution<double> dist(-1.0, 1.0);
    
    for (int i = 0; i < size; i++) {
        vector[i] = make_hipDoubleComplex(dist(gen), dist(gen));
    }
}

// Define constants following the driver pattern
const int dsymv_lower_bs = 32;  // Reduced from 64 to 32 for complex numbers
const int dsymv_lower_ty = 4;
const int dsymv_lower_by = 4;

// Wrapper function for custom kernel launch
void launchCustomHemv(int n, hipDoubleComplex alpha, hipDoubleComplex *d_A, int lda, 
                     hipDoubleComplex *d_x, int incx, hipDoubleComplex beta, 
                     hipDoubleComplex *d_y, int incy) {
    
    // Handle negative increments
    if(incx < 0) d_x -= (n-1) * incx;
    if(incy < 0) d_y -= (n-1) * incy;
    
    // Configuration params
    const int dsymv_bs = dsymv_lower_bs;
    const int thread_x = dsymv_bs;
    const int thread_y = dsymv_lower_ty;
    const int elements_per_thread = (dsymv_bs/(2*thread_y));
    
    int mod = n % dsymv_bs;
    int blocks = n / dsymv_bs + (mod != 0);
    dim3 dimBlock(thread_x, thread_y);
    dim3 dimGrid(blocks, 1);
    dim3 dimGrid_(blocks, dsymv_lower_by);
    
    if(mod == 0) {
        syhemvl_generic_d<hipDoubleComplex, dsymv_bs, thread_x, thread_y, elements_per_thread>
            <<<dimGrid, dimBlock>>>(n, alpha, d_A, lda, d_x, incx, beta, d_y, incy, 0);
        syhemvl_generic_nd<hipDoubleComplex, dsymv_bs, thread_x, thread_y, elements_per_thread>
            <<<dimGrid_, dimBlock>>>(n, alpha, d_A, lda, d_x, incx, beta, d_y, incy, 0);
    } else {
        syhemvl_generic_d<hipDoubleComplex, dsymv_bs, thread_x, thread_y, elements_per_thread>
            <<<dimGrid, dimBlock>>>(n, alpha, d_A, lda, d_x, incx, beta, d_y, incy, mod);
        syhemvl_generic_nd<hipDoubleComplex, dsymv_bs, thread_x, thread_y, elements_per_thread>
            <<<dimGrid_, dimBlock>>>(n, alpha, d_A, lda, d_x, incx, beta, d_y, incy, mod);
    }
}

int main() {
    const int N = 4096;
    const hipDoubleComplex alpha = make_hipDoubleComplex(1.0, 0.0);
    const hipDoubleComplex beta = make_hipDoubleComplex(0.0, 0.0);
    const int incx = 1;
    const int incy = 1;
    const int warmup_runs = 5;
    const int benchmark_runs = 20;
    
    // Host memory allocation
    std::vector<hipDoubleComplex> h_A(N * N);
    std::vector<hipDoubleComplex> h_x(N);
    std::vector<hipDoubleComplex> h_y_cublas(N);
    std::vector<hipDoubleComplex> h_y_custom(N);
    std::vector<hipDoubleComplex> h_y_original(N);
    
    // Initialize matrices and vectors
    initializeHermitianMatrix(h_A.data(), N);
    initializeComplexVector(h_x.data(), N);
    initializeComplexVector(h_y_original.data(), N);
    
    // Copy original y for both tests
    std::copy(h_y_original.begin(), h_y_original.end(), h_y_cublas.begin());
    std::copy(h_y_original.begin(), h_y_original.end(), h_y_custom.begin());
    
    // Device memory allocation
    hipDoubleComplex *d_A, *d_x, *d_y_cublas, *d_y_custom;
    CHECK_CUDA(hipMalloc(&d_A, N * N * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc(&d_x, N * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc(&d_y_cublas, N * sizeof(hipDoubleComplex)));
    CHECK_CUDA(hipMalloc(&d_y_custom, N * sizeof(hipDoubleComplex)));
    
    // Copy data to device
    CHECK_CUDA(hipMemcpy(d_A, h_A.data(), N * N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_x, h_x.data(), N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y_cublas, h_y_cublas.data(), N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    CHECK_CUDA(hipMemcpy(d_y_custom, h_y_custom.data(), N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
    
    // cuBLAS setup
    hipblasHandle_t handle;
    CHECK_CUBLAS(hipblasCreate(&handle));
    
    // CUDA events for timing
    hipEvent_t start, stop;
    CHECK_CUDA(hipEventCreate(&start));
    CHECK_CUDA(hipEventCreate(&stop));
    
    // Warm-up cuBLAS
    std::cout << "Warming up cuBLAS..." << std::endl;
    for (int i = 0; i < warmup_runs; i++) {
        CHECK_CUBLAS(hipblasZhemv(handle, HIPBLAS_FILL_MODE_LOWER, N, &alpha, d_A, N, d_x, incx, &beta, d_y_cublas, incy));
    }
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Benchmark cuBLAS
    std::cout << "Benchmarking cuBLAS zhemv..." << std::endl;
    float cublas_total_time = 0.0f;
    
    for (int i = 0; i < benchmark_runs; i++) {
        // Reset y vector
        CHECK_CUDA(hipMemcpy(d_y_cublas, h_y_original.data(), N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        
        CHECK_CUDA(hipEventRecord(start));
        CHECK_CUBLAS(hipblasZhemv(handle, HIPBLAS_FILL_MODE_LOWER, N, &alpha, d_A, N, d_x, incx, &beta, d_y_cublas, incy));
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        
        float milliseconds;
        CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
        cublas_total_time += milliseconds;
    }
    float cublas_avg_time = cublas_total_time / benchmark_runs;
    
    // Copy cuBLAS result back to host for verification
    CHECK_CUDA(hipMemcpy(h_y_cublas.data(), d_y_cublas, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    
    // Warm-up custom kernel
    std::cout << "Warming up custom kernel..." << std::endl;
    int mod = N % dsymv_lower_bs;
    int blocks = N / dsymv_lower_bs + (mod != 0);
    
    // Check shared memory requirements
    size_t shmem_diag = dsymv_lower_bs * dsymv_lower_bs * sizeof(hipDoubleComplex) + 
                        dsymv_lower_bs * sizeof(hipDoubleComplex) + 
                        dsymv_lower_bs * (2 * dsymv_lower_ty) * sizeof(hipDoubleComplex);
    size_t shmem_ndiag = dsymv_lower_bs * (dsymv_lower_bs/2) * sizeof(hipDoubleComplex) + 
                         dsymv_lower_bs * (2 * dsymv_lower_ty) * sizeof(hipDoubleComplex) + 
                         dsymv_lower_bs * sizeof(hipDoubleComplex);
    
    std::cout << "Kernel configuration:" << std::endl;
    std::cout << "  Matrix size: " << N << "x" << N << std::endl;
    std::cout << "  Block size (dsymv_bs): " << dsymv_lower_bs << std::endl;
    std::cout << "  Mod: " << mod << std::endl;
    std::cout << "  Blocks: " << blocks << std::endl;
    std::cout << "  Thread block: (" << dsymv_lower_bs << ", " << dsymv_lower_ty << ")" << std::endl;
    std::cout << "  Diagonal grid: (" << blocks << ", 1)" << std::endl;
    std::cout << "  Non-diagonal grid: (" << blocks << ", " << dsymv_lower_by << ")" << std::endl;
    std::cout << "  Elements per thread: " << (dsymv_lower_bs/(2*dsymv_lower_ty)) << std::endl;
    std::cout << "  Shared memory (diag): " << shmem_diag << " bytes" << std::endl;
    std::cout << "  Shared memory (ndiag): " << shmem_ndiag << " bytes" << std::endl;
    std::cout << "  Using " << (mod == 0 ? "special" : "generic") << " kernels (Lower Hermitian)" << std::endl;
    
    for (int i = 0; i < warmup_runs; i++) {
        launchCustomHemv(N, alpha, d_A, N, d_x, incx, beta, d_y_custom, incy);
        CHECK_CUDA(hipGetLastError()); // Check for kernel errors
    }
    CHECK_CUDA(hipDeviceSynchronize());
    
    // Benchmark custom kernel
    std::cout << "Benchmarking custom hemv kernel..." << std::endl;
    float custom_total_time = 0.0f;
    
    for (int i = 0; i < benchmark_runs; i++) {
        // Reset y vector
        CHECK_CUDA(hipMemcpy(d_y_custom, h_y_original.data(), N * sizeof(hipDoubleComplex), hipMemcpyHostToDevice));
        
        CHECK_CUDA(hipEventRecord(start));
        launchCustomHemv(N, alpha, d_A, N, d_x, incx, beta, d_y_custom, incy);
        CHECK_CUDA(hipGetLastError()); // Check for kernel errors
        CHECK_CUDA(hipEventRecord(stop));
        CHECK_CUDA(hipEventSynchronize(stop));
        
        float milliseconds;
        CHECK_CUDA(hipEventElapsedTime(&milliseconds, start, stop));
        custom_total_time += milliseconds;
    }
    float custom_avg_time = custom_total_time / benchmark_runs;
    
    // Copy custom kernel result back to host for verification
    CHECK_CUDA(hipMemcpy(h_y_custom.data(), d_y_custom, N * sizeof(hipDoubleComplex), hipMemcpyDeviceToHost));
    
    // Debug: Print first few elements for comparison
    std::cout << "\nDebug - First 10 elements comparison:" << std::endl;
    std::cout << "Index\tcuBLAS\t\t\tCustom\t\t\tDiff" << std::endl;
    for (int i = 0; i < 10; i++) {
        double diff_real = std::abs(hipCreal(h_y_cublas[i]) - hipCreal(h_y_custom[i]));
        double diff_imag = std::abs(hipCimag(h_y_cublas[i]) - hipCimag(h_y_custom[i]));
        double diff = std::sqrt(diff_real * diff_real + diff_imag * diff_imag);
        std::cout << i << "\t" << std::scientific << std::setprecision(6) 
                  << hipCreal(h_y_cublas[i]) << "+" << hipCimag(h_y_cublas[i]) << "i\t" 
                  << hipCreal(h_y_custom[i]) << "+" << hipCimag(h_y_custom[i]) << "i\t" 
                  << diff << std::endl;
    }
    
    // Performance results
    std::cout << "\n=== Performance Results ===" << std::endl;
    std::cout << "Matrix size: " << N << "x" << N << std::endl;
    std::cout << std::endl;
    
    std::cout << "Timing Results (average of " << benchmark_runs << " runs):" << std::endl;
    std::cout << std::fixed << std::setprecision(6);
    std::cout << "cuBLAS zhemv:     " << cublas_avg_time << " ms" << std::endl;
    std::cout << "Custom kernel:    " << custom_avg_time << " ms" << std::endl;
    std::cout << std::endl;
    
    // Calculate speedup
    float speedup = cublas_avg_time / custom_avg_time;
    std::cout << "Speedup: " << std::setprecision(5) << speedup << "x ";
    if (speedup > 1.0f) {
        std::cout << "(Custom kernel is faster)" << std::endl;
    } else {
        std::cout << "(cuBLAS is faster)" << std::endl;
    }
    std::cout << std::endl;
    
    // Verification
    std::cout << "Verification:" << std::endl;
    bool results_match = compareResults(h_y_cublas.data(), h_y_custom.data(), N);
    if (results_match) {
        std::cout << "✓ Results match (within numerical precision)" << std::endl;
    } else {
        std::cout << "✗ Results do not match!" << std::endl;
    }
    
    // Cleanup
    CHECK_CUBLAS(hipblasDestroy(handle));
    CHECK_CUDA(hipEventDestroy(start));
    CHECK_CUDA(hipEventDestroy(stop));
    CHECK_CUDA(hipFree(d_A));
    CHECK_CUDA(hipFree(d_x));
    CHECK_CUDA(hipFree(d_y_cublas));
    CHECK_CUDA(hipFree(d_y_custom));
    
    return 0;
}